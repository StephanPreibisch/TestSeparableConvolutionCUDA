#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */



#include <assert.h>
#include <hip/hip_runtime_api.h>
#include "convolutionSeparable_common.h"



////////////////////////////////////////////////////////////////////////////////
// Convolution kernel storage
////////////////////////////////////////////////////////////////////////////////
__constant__ float c_Kernel[KERNEL_LENGTH];

extern "C" void setConvolutionKernel(float *h_Kernel)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), h_Kernel, KERNEL_LENGTH * sizeof(float));
}


////////////////////////////////////////////////////////////////////////////////
// Row convolution filter
////////////////////////////////////////////////////////////////////////////////

// how many threads per block in x (total num threads: x*y)
#define   ROWS_BLOCKDIM_X 16

// how many threads per block in y
#define   ROWS_BLOCKDIM_Y 4

// how many pixels in x are convolved by each thread
#define ROWS_RESULT_STEPS 8

// these are the border pixels (loaded to support the kernel width for processing)
// the effective border width is ROWS_HALO_STEPS * ROWS_BLOCKDIM_X, which has to be
// larger or equal to the kernel radius to work
#define   ROWS_HALO_STEPS 1

__global__ void convolution2dRowsKernel(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
)
{
    // shared memory of pixels for all threads (n=ROWS_BLOCKDIM_X * ROWS_BLOCKDIM_Y) of one block
    // note: up to Compute capability 3.5 this is 48kb
    //
    // in y: the number of threads (ROWS_BLOCKDIM_Y)
    // in x: the number of threads (ROWS_BLOCKDIM_X) * (ROWS_RESULT_STEPS=Number of processed pixels + twice the ROW_HALO_STEPS (whatever the fuck this is) )
    __shared__ float s_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

    // Offset (in the input and output array!) to the left halo edge relative to the current block and thread that is processed, 
    // i.e. which pixels are we copying in this thread?
    //
    // blockIdx.x and blockIdx.y give us an index for the current block, so the pixel coordinates in x and y result from the number of threads per block,
    // or many also be called the blocksize ROWS_BLOCKDIM_X and ROWS_BLOCKDIM_Y
    //
    // threadIdx.x and threadIdx.y give use the corresponding index of the current thread within the block, so 
    //
    // This is simple in y, the line we are in is just the current block times the blocksize (#of threads in y) plus the current thread
    // It is a little more complicated in x. The threads are set one after another left of the actual data that is processed, and are
    // increased in ROWS_BLOCKDIM_X steps to fill up the shared memory. The amount of space next to the data that is convolved is determined
    // by ROWS_HALO_STEPS * ROWS_BLOCKDIM_X, so in the default version it is 16 pixels, which is the maximally supported kernel radius.
    const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;

    // set the input and output arrays to the right offset (actually the output is not at the right offset, but this is corrected later)
    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    // Load main data
    // Start copying after the ROWS_HALO_STEPS, only the original data that will be convolved
#pragma unroll

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = d_Src[i * ROWS_BLOCKDIM_X];
    }

    // Load left halo
    // If the data fetched is outside of the image (note: baseX can be <0 for the first block) , use a zero-out of bounds strategy
#pragma unroll

    for (int i = 0; i < ROWS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX >= -i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
    }

    //Load right halo
#pragma unroll

    for (int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
    }

    //Compute and store results
    __syncthreads();
#pragma unroll

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        float sum = 0;

#pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
            sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j];
        }

        d_Dst[i * ROWS_BLOCKDIM_X] = sum;
    }
}

extern "C" void convolution2dRowsGPU(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH
)
{
    assert(ROWS_BLOCKDIM_X * ROWS_HALO_STEPS >= KERNEL_RADIUS);
    assert(imageW % (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X) == 0);
    assert(imageH % ROWS_BLOCKDIM_Y == 0);

    dim3 blocks(imageW / (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X), imageH / ROWS_BLOCKDIM_Y);
    dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);

    convolution2dRowsKernel<<<blocks, threads>>>(
        d_Dst,
        d_Src,
        imageW,
        imageH,
        imageW
    );
    getLastCudaError("convolutionRowsKernel() execution failed\n");
}



////////////////////////////////////////////////////////////////////////////////
// Column convolution filter
////////////////////////////////////////////////////////////////////////////////
#define   COLUMNS_BLOCKDIM_X 16
#define   COLUMNS_BLOCKDIM_Y 8
#define COLUMNS_RESULT_STEPS 8
#define   COLUMNS_HALO_STEPS 1

__global__ void convolution2dColumnsKernel(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
)
{
    __shared__ float s_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS + 2 * COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + 1];

    //Offset to the upper halo edge
    const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
    const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //Main data
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = d_Src[i * COLUMNS_BLOCKDIM_Y * pitch];
    }

    //Upper halo
#pragma unroll

    for (int i = 0; i < COLUMNS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
    }

    //Lower halo
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y]= (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
    }

    //Compute and store results
    __syncthreads();
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
    {
        float sum = 0;
#pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
            sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j];
        }

        d_Dst[i * COLUMNS_BLOCKDIM_Y * pitch] = sum;
    }
}

extern "C" void convolution2dColumnsGPU(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH
)
{
    assert(COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS >= KERNEL_RADIUS);
    assert(imageW % COLUMNS_BLOCKDIM_X == 0);
    assert(imageH % (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y) == 0);

    dim3 blocks(imageW / COLUMNS_BLOCKDIM_X, imageH / (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y));
    dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);

    convolution2dColumnsKernel<<<blocks, threads>>>(
        d_Dst,
        d_Src,
        imageW,
        imageH,
        imageW
    );
    getLastCudaError("convolutionColumnsKernel() execution failed\n");
}

//
// 3d implementation
//

__global__ void convolution3dRowsKernel(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int imageD,
    int pitchX,
    int pitchY
)
{
    __shared__ float s_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

    const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;
    const int baseZ = blockIdx.z;

    // set the input and output arrays to the right offset (actually the output is not at the right offset, but this is corrected later)
    d_Src += baseZ * pitchY * pitchX + baseY * pitchX + baseX;
    d_Dst += baseZ * pitchY * pitchX + baseY * pitchX + baseX;

    // Load main data
    // Start copying after the ROWS_HALO_STEPS, only the original data that will be convolved
#pragma unroll

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = d_Src[i * ROWS_BLOCKDIM_X];
    }

    // Load left halo
    // If the data fetched is outside of the image (note: baseX can be <0 for the first block) , use a zero-out of bounds strategy
#pragma unroll

    for (int i = 0; i < ROWS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX >= -i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
    }

    //Load right halo
#pragma unroll

    for (int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
    }

    //Compute and store results
    __syncthreads();
#pragma unroll

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        float sum = 0;

#pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
            sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j];
        }

        d_Dst[i * ROWS_BLOCKDIM_X] = sum;
    }
}

extern "C" void convolution3dRowsGPU(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int imageD
)
{
    assert(ROWS_BLOCKDIM_X * ROWS_HALO_STEPS >= KERNEL_RADIUS);
    assert(imageW % (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X) == 0);
    assert(imageH % ROWS_BLOCKDIM_Y == 0);

    dim3 blocks(imageW / (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X), imageH / ROWS_BLOCKDIM_Y, imageD);
    dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);

    convolution3dRowsKernel<<<blocks, threads>>>(
        d_Dst,
        d_Src,
        imageW,
        imageH,
        imageD,
        imageW,
        imageH
    );
    getLastCudaError("convolution3dRowsKernel() execution failed\n");
}

__global__ void convolution3dColumnsKernel(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int imageD,
    int pitchX,
    int pitchY
)
{
    __shared__ float s_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS + 2 * COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + 1];

    //Offset to the upper halo edge
    const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
    const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
    const int baseZ = blockIdx.z + threadIdx.z;

    d_Src += baseZ * pitchY * pitchX + baseY * pitchX + baseX;
    d_Dst += baseZ * pitchY * pitchX + baseY * pitchX + baseX;

    //Main data
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = d_Src[i * COLUMNS_BLOCKDIM_Y * pitchX];
    }

    //Upper halo
#pragma unroll

    for (int i = 0; i < COLUMNS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitchX] : 0;
    }

    //Lower halo
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y]= (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitchX] : 0;
    }

    //Compute and store results
    __syncthreads();
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
    {
        float sum = 0;
#pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
            sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j];
        }

        d_Dst[i * COLUMNS_BLOCKDIM_Y * pitchX] = sum;
    }
}

extern "C" void convolution3dColumnsGPU(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int imageD
)
{
    assert(COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS >= KERNEL_RADIUS);
    assert(imageW % COLUMNS_BLOCKDIM_X == 0);
    assert(imageH % (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y) == 0);

    dim3 blocks(imageW / COLUMNS_BLOCKDIM_X, imageH / (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y), imageD);
    dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y, 1);

    convolution3dColumnsKernel<<<blocks, threads>>>(
        d_Dst,
        d_Src,
        imageW,
        imageH,
        imageD,
        imageW,
        imageH
    );
    getLastCudaError("convolution3dColumnsKernel() execution failed\n");
}

#define   DEPTH_BLOCKDIM_X 16
#define   DEPTH_BLOCKDIM_Z 8
#define DEPTH_RESULT_STEPS 8
#define   DEPTH_HALO_STEPS 1

__global__ void convolution3dDepthKernel(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int imageD,
    int pitchX,
    int pitchY
)
{
	// here it is [x][z], we leave out y as it has a size of 1
    __shared__ float s_Data[DEPTH_BLOCKDIM_X][(DEPTH_RESULT_STEPS + 2 * DEPTH_HALO_STEPS) * DEPTH_BLOCKDIM_Z + 1];

    //Offset to the upper halo edge
    const int baseX = blockIdx.x * DEPTH_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y + threadIdx.y;
    const int baseZ = (blockIdx.z * DEPTH_RESULT_STEPS - DEPTH_HALO_STEPS) * DEPTH_BLOCKDIM_Z + threadIdx.z;

    d_Src += baseZ * pitchY * pitchX + baseY * pitchX + baseX;
    d_Dst += baseZ * pitchY * pitchX + baseY * pitchX + baseX;

    //Main data
#pragma unroll

    for (int i = DEPTH_HALO_STEPS; i < DEPTH_HALO_STEPS + DEPTH_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.z + i * DEPTH_BLOCKDIM_Z] = d_Src[i * DEPTH_BLOCKDIM_Z * pitchX * pitchY];
    }

    //Upper halo
#pragma unroll

    for (int i = 0; i < DEPTH_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.z + i * DEPTH_BLOCKDIM_Z] = (baseZ >= -i * DEPTH_BLOCKDIM_Z) ? d_Src[i * DEPTH_BLOCKDIM_Z * pitchX * pitchY] : 0;
    }

    //Lower halo
#pragma unroll

    for (int i = DEPTH_HALO_STEPS + DEPTH_RESULT_STEPS; i < DEPTH_HALO_STEPS + DEPTH_RESULT_STEPS + DEPTH_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * DEPTH_BLOCKDIM_Z]= (imageD - baseZ > i * DEPTH_BLOCKDIM_Z) ? d_Src[i * DEPTH_BLOCKDIM_Z * pitchX * pitchY] : 0;
    }

    //Compute and store results
    __syncthreads();
#pragma unroll

    for (int i = DEPTH_HALO_STEPS; i < DEPTH_HALO_STEPS + DEPTH_RESULT_STEPS; i++)
    {
        float sum = 0;
#pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
            sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.x][threadIdx.z + i * DEPTH_BLOCKDIM_Z + j];
        }

        d_Dst[i * DEPTH_BLOCKDIM_Z * pitchX * pitchY] = sum;
    }
}

extern "C" void convolution3dDepthGPU(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int imageD
)
{
    assert(DEPTH_BLOCKDIM_Z * DEPTH_HALO_STEPS >= KERNEL_RADIUS);
    assert(imageW % DEPTH_BLOCKDIM_X == 0);
    assert(imageD % (DEPTH_RESULT_STEPS * DEPTH_BLOCKDIM_Z) == 0);

    dim3 blocks(imageW / DEPTH_BLOCKDIM_X, imageH, imageD/ (DEPTH_RESULT_STEPS * DEPTH_BLOCKDIM_Z) );
    dim3 threads(DEPTH_BLOCKDIM_X, 1, DEPTH_BLOCKDIM_Z);

    convolution3dDepthKernel<<<blocks, threads>>>(
        d_Dst,
        d_Src,
        imageW,
        imageH,
        imageD,
        imageW,
        imageH
    );
    getLastCudaError("convolution3dDepthKernel() execution failed\n");
}

